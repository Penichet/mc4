
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>

using namespace std;


__global__ void global_reduce_kernel(int* d_out, int* d_in, int size)
{   //indices
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    
    // do reduction in global mem
    for (unsigned int cap = blockDim.x / 2; cap > 0; cap >>= 1) 
    {
        //only compute if on lower portion of block
        if (tid < cap)
        {

            //if thread out of range or threads comp out of range, do nothing
            if(myId >= size || myId + cap >=size){
                //do nothing
            }
            else{
                // store minimum only between two valid elements in lower portion
                d_in[myId] = min(d_in[myId], d_in[myId + cap]);
            }
            
        }
        //wait for all threads to complete
        __syncthreads();
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = d_in[myId];
    }
}

void reduce(int* d_out, int* d_intermediate, int* d_in, int size)
{
    /*int threads_num, numProcs;
    cudaDeviceGetAttribute(&threads_num, cudaDevAttrMaxThreadsPerMultiProcessor, 0);
    printf("max threads per mp: %d\n",  threads_num);
    cudaDeviceGetAttribute(&numProcs,cudaDevAttrMultiProcessorCount, 0);
    printf("mp count: %d\n", numProcs);*/

    const int maxThreadsPerBlock = 512;
    int threads = maxThreadsPerBlock;

    //ceiling of blocks required
    int blocks = (size / maxThreadsPerBlock)+1; 
    

    global_reduce_kernel<<<blocks, threads >>>(d_intermediate, d_in, size);

    // now we're down to one block left, so reduce it
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;

    // set threads to multiple of two greater than or equal to size 
    int mult = 1;
    while (mult < threads) mult *= 2;

    //launch kernel with multiple of 2 threads, and size equal to number of valid entries
    global_reduce_kernel<<<blocks, mult >>>(d_out, d_intermediate, threads);
    
}


int main() {
    vector<int> arr;
    string line;
    ifstream myfile("inp.txt");
    if (myfile.is_open())
    {
        //gets next int
        while (getline(myfile, line, ','))
        {
            arr.push_back(stoi(line, nullptr));
        }
        myfile.close();
    }
    else cout << "Unable to open file";




    //timing stuff
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //allocated device memory
    int *d_arr, *d_out, *d_intermediate;
    hipMalloc((void**)&d_arr, arr.size() * sizeof(int));
    hipMalloc((void**)&d_out, sizeof(int));
    hipMalloc((void**)&d_intermediate, arr.size() * sizeof(int));

    // treat pointer to start of vector as array pointer
    hipMemcpy(d_arr, &arr[0], arr.size() * sizeof(int), hipMemcpyHostToDevice);

    //run reduce operation
    hipEventRecord(start, 0);
    reduce(d_out, d_intermediate, d_arr, arr.size());
    hipEventRecord(stop, 0);
    
    //wait for it to finish
    hipDeviceSynchronize();

    //store answer on host
    int ans;
    hipMemcpy(&ans, d_out, sizeof(int), hipMemcpyDeviceToHost);

    //find time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    //print stuff
    cout << "minimum entry found: " << ans << endl;
    cout << "elapsted time: " << elapsedTime << endl;

    hipFree(d_arr);
    hipFree(d_intermediate);
    hipFree(d_arr);

    return 0;
}




