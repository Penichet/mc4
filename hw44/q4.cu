
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
using namespace std;

//https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
//child kernel for prefix scan - NOT SURE IF INCLUSIVE OR EXCLUSIVE
//__device__ void prefix_scan(int* d_out, int* flag, int* temp,  int size) {
__global__ void prefix_scan(int* d_out, int* flag, int size) {
    //TODO: need to adjust size to multiple of 2 and also resize temp array 

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    //need to initialize all in temp to 0?
    
    d_out[myId] = flag[myId]; // load inputs into memory 

    for (int h = 1; h < size; h *= 2) {
        int index = h*2* myId + h * 2 - 1;
        if(index < size) { //check if myid +step is smaller than size
            d_out[index] += d_out[h*2* myId + h - 1];

            //NEED TO PAD INPUT ARRAY
        }
        __syncthreads();
    }

    //B[n - 1] = 0;
    if (myId == 0) {
        d_out[size - 1] = 0;
    } // clear the last element
    __syncthreads();

    //        leftval = B[i + h -1]--------------------------
    //        B[i + (h) - 1] = B[i + (h*2) -1] ---------------------------
    //        B[i + (h*2) - 1] = B[i + (h*2) -1] + leftVal; ------------------------------
    //0,1,3,6,10,15,21,28, 36,45,55,66,78,91,105,120 - EXPECTED
                                          
    for (int h = size/2; h > 0; h /= 2) {
        int index = h*2*myId + (h * 2) - 1; 
        int right = h*2*myId + (h*1) - 1;
        if (index < size) { 
            int leftVal = d_out[right];
            d_out[right] = d_out[index];
            d_out[index] += leftVal;
        }
        __syncthreads();
    }

}



__global__ void global_bucket_sort(int* d_out, int* d_in, int* flags, int* scan,int* temp, int size) {
    //indices
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ int numFalse;
    //int tid = threadIdx.x;
    int mask = 1;
    
    if (myId < size) {
        // for every bit, increase mask 1 bit 
        for (int i = 0; i < 10; i++, mask <<= 1) {

            //MAP 0s as 1 in flags array
            if ((mask & d_in[myId])==0) {
                //if (tid == 0) printf("masking %d with %d, returned 0", d_in[myId], mask);
                flags[myId] = 1;
            }
            //synch all threads
            __syncthreads();
            printf("we haven't failed yet");
            //run prefix scan on 0s - should be inlined??
            //prefix_scan(scan, flags, size);
            numFalse = scan[size - 1] + flags[size - 1]; // number of falses total
            __syncthreads();
            //scan now holds results of scan from flags

            int t = myId - scan[myId] + numFalse; // true index for all
            __syncthreads();

            //if bit true, use t index
            if (mask & d_in[myId] != 0) {
                d_out[t] = d_in[myId];
            }
            else { //if bit false, use scan index aka f index
                d_out[scan[myId]] = d_in[myId];
            }
            __syncthreads();

            //copy to d_in to redo next layer
            d_in[myId] = d_out[myId];
            __syncthreads();

            //clear used arrays
            flags[myId] = 0;
            scan[myId] = 0;
            __syncthreads();
            //run next bit mask
        }
    }
}


void bucket(int* d_out, int* d_in, int* flags, int* scan, int size) {
    const int maxThreadsPerBlock = 512;
    int threads = maxThreadsPerBlock;
    int blocks = (size / maxThreadsPerBlock) + 1;
    //global_bucket_sort <<<blocks, threads>>> (d_out, d_in, flags, scan, temp, size);

    // testing scan///////////////////////////////////////////
    int* temp;
    vector<int> temparr;
    for (int i = 1; i <= 8192; i++) {
        temparr.push_back(1);
    }
    hipMalloc((void**)&temp, temparr.size() * sizeof(int));
    hipMemcpy(temp, &temparr[0], temparr.size() * sizeof(int), hipMemcpyHostToDevice);
    printf("starting prefix scan\n");
    // more than 1024 is messed up, prolly bc 1024 threads per block
    int tempsize = (8192 / 512);
    prefix_scan <<<tempsize, 512 >>> (scan, temp, temparr.size());
    int* ans_scan = (int*)malloc(sizeof(int) * temparr.size());
    hipMemcpy(ans_scan, scan, sizeof(int) * temparr.size(), hipMemcpyDeviceToHost);
    cout << ans_scan[0];
    for (int i = 1; i < 8192; i++) {
        cout << "," << ans_scan[i];
    }
    ///////////////////////////////

}

int main(){
    vector<int> arr;
    string line;
    ifstream myfile("inp.txt");
    if (myfile.is_open())
    {
        //gets next int
        while (getline(myfile, line, ','))
        {
            arr.push_back(stoi(line, nullptr));
        }
        myfile.close();
    }
    else cout << "Unable to open file";
    //Array A is now accessible as arr

    //allocate device memory
    int* d_arr, *d_out, *scan, *flags;
    hipMalloc((void**)&d_arr, arr.size() * sizeof(int));
    hipMalloc((void**)&d_out, arr.size() * sizeof(int));
    hipMalloc((void**)&flags, arr.size() * sizeof(int));
    hipMalloc((void**)&scan, arr.size() * sizeof(int));
    //cudaMalloc((void**)&temp, arr.size() * sizeof(int));


    // treat pointer to start of vector as array pointer
    hipMemcpy(d_arr, &arr[0], arr.size() * sizeof(int), hipMemcpyHostToDevice);

    //Cuda Events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //run reduce operation
    hipEventRecord(start, 0);
    bucket(d_out, d_arr, flags, scan, arr.size());
    hipEventRecord(stop, 0);

    //copy results
    int* ans_arr = (int*)malloc(sizeof(int) * arr.size());
    hipMemcpy(ans_arr, d_out, sizeof(int) * arr.size(), hipMemcpyDeviceToHost);

    //output to file
    ofstream outfile2("q4.txt");
    if (outfile2.is_open())
    {
        //avoid comma at end of string
        outfile2 << ans_arr[0];
        for (int i = 1; i < arr.size(); i++) {
            outfile2 << "," << ans_arr[i];
        }

        outfile2.close();
    }
}